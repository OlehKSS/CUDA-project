#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#include <math.h>
#include <ctime>
#include <iostream>




__global__ void BitonicMergeSort(float * d_output, float * d_input, int subarray_size)
{
    extern __shared__ float shared_data[];
    int index  = blockDim.x * blockIdx.x + threadIdx.x;
	double portions = log2(double(subarray_size))-1;

    shared_data[index] = d_input[index];
    __syncthreads();

    for (short portion = 0; portion <= portions; portion++)
    {
        short offset = 1<<portion;
        short threads_in_box = offset<<1;
        // calculated at the beginning of each portion
        //int boxI = index % (threads_in_box + (blockDim.x * blockIdx.x));
		int boxI = threadIdx.x/threads_in_box;
        for (short subportion = portion; subportion >= 0; subportion--)
        {
            offset = 1<<subportion;
            threads_in_box = offset<<1;
            int arrow_bottom = index % threads_in_box;

            if (((boxI + 1) % 2) == 1) {
                // top down
                if (arrow_bottom < offset) {
                    float temp = shared_data[index];
                    if (shared_data[index+offset] < temp) {
                        shared_data[index] = shared_data[index+offset];
                        shared_data[index+offset] = temp;
                    }
                }
            } else {
                // bottom up
                if (arrow_bottom >= offset) {
                    float temp = shared_data[index];
                    if (shared_data[index-offset] < temp) {
                        shared_data[index] = shared_data[index-offset];
                        shared_data[index-offset] = temp;
                    }
                }
            }
            __syncthreads();
        }
    }

    d_output[index] = shared_data[index];

}


int main(int argc, char **argv)
{
	int n_el = 8192;

    int ARRAY_SIZE = pow(2, ceil(log(n_el)/log(2)));;
    int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);


    // generate the input array on the host
    float *h_input = new float[ARRAY_SIZE];
    float *h_output = new float[ARRAY_SIZE];
    for(int i = 0; i < n_el; i++) {
        // generate random float in [0, 999]
        //h_input[i] = (float)rand()/(float)RAND_MAX;
		h_input[i] = rand()%10+1;
    }
	for(int i = n_el; i < ARRAY_SIZE; i++) {
        // generate random float in [0, 999]
        //h_input[i] = (float)rand()/(float)RAND_MAX;
		h_input[i] = 0;
    }

    // declare GPU memory pointers
    float * d_input, * d_output;

    // allocate GPU memory
    hipMalloc((void **) &d_input, ARRAY_BYTES);
    hipMalloc((void **) &d_output, ARRAY_BYTES);

    

    // launch the kernel
	int threads_per_block = ARRAY_SIZE;
	int num_blocks = int((ARRAY_SIZE-1)/1024) + 1;
	if(ARRAY_SIZE > 1024)
		threads_per_block = 1024;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// transfer the input array to the GPU
    hipMemcpy(d_input, h_input, ARRAY_BYTES, hipMemcpyHostToDevice);

	hipEventRecord(start);

	int subarray_size = 256;

    BitonicMergeSort<<<32, subarray_size, ARRAY_SIZE * sizeof(float)>>>(d_output, d_input, subarray_size);

	hipEventRecord(stop);

	// copy back the sum from GPU
    hipMemcpy(h_output, d_output, ARRAY_BYTES, hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

	//for(int i = 0; i < ARRAY_SIZE; i++){
	//	printf("%f \n",h_input[i]);
	//}
	//printf("\n\n");
	float *final_array = h_output+(ARRAY_SIZE-n_el);

	for(int i = 0; i < n_el; i++){
		printf("%.0f ",final_array[i]);
	}

	std::cout << "Merge sort, GPU time elapsed (millisec) " << milliseconds << std::endl;
	getchar();

    // free GPU memory allocation
	delete[] h_input;
	delete[] h_output;
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}